//--------------------------------------------------------------------------
// Project: 
// Read GPU temepratures on a CUDA-enabled system.
// Bypass the need for 3rd party libraries.
// Insert into your code as desired. 
//
// Prerequisites: 
// Must have installed the CUDA toolkit.
// Must be running on a UNIX machine
//
// Independent testing info:
// Compile on commandline: nvcc read_temps.cu -o test
// run on commandline: ./test
//
// Author: Jordan Bonilla
// Date  : April 2016
// License: All rights Reserved. See LICENSE.txt
//--------------------------------------------------------------------------

#include <cstdio> // printf
#include <stdlib.h> // popen, pclose, atoi, fread
#include <hip/hip_runtime.h> // cudaGetDeviceCount

// Read temperatures of all connected NVIDIA GPUs
void read_temps() 
{
    // Get the number of GPUs on this machine
    int num_devices;
    hipGetDeviceCount(&num_devices);
    if(num_devices == 0) {
      printf("No NVIDIA GPUs detected\n");
        return;
    }
    // Read GPU info into buffer "output" using the "nvidia-smi" command
    const unsigned int MAX_BYTES = 10000;
    char output[MAX_BYTES];
    FILE *fp = popen("nvidia-smi &> /dev/null", "r");
    fread(output, sizeof(char), MAX_BYTES, fp);
    pclose(fp);
    // array to hold GPU temperatures
    int * temperatures = new int[num_devices];
    // parse output for temperatures using knowledge of "nvidia-smi" output format
    int i = 0;
    unsigned int num_temps_parsed = 0;
    while(output[i] != '\0') {
        if(output[i] == '%') {
            unsigned int temp_begin = i + 1;
            while(output[i] != 'C') {
                 ++i;
            }
            unsigned int temp_end = i;
            char this_temperature[32];
                        // Read in the characters cooresponding to this temperature
                        for(int j = 0; j < temp_end - temp_begin; ++j) {
                            this_temperature[j] = output[temp_begin + j];
                        }
                        this_temperature[temp_end - temp_begin + 1] = '\0';
            temperatures[num_temps_parsed] = atoi(this_temperature);
            num_temps_parsed++;
        }
        ++i;
    }
    for (int i = 0; i < num_devices; i++) 
    {
      printf("GPU %d temperature: %d C\n", i, temperatures[i]);
    }

    // Free memory and return
    delete(temperatures);
    return;
}

int main(int argc, char **argv) {
    read_temps();
    return 0;
}
